#include "hip/hip_runtime.h"
#include "Vision.h"
#include "MathUtils.h"
#include "RobotStateParser.h"

#include <opencv2/calib3d.hpp>
#include <iostream>
#include <opencv2/features2d.hpp>

Vision::Vision(CameraReceiver &overheadCam)
    : overheadCam(overheadCam),
      opticalFlow(),
      opponentOpticalFlow()
{
    // add 2 robot trackers
    robotTrackers.push_back(RobotTracker(cv::Point2f(0,0)));
    robotTrackers.push_back(RobotTracker(cv::Point2f(10000, 10000)));

}

void Vision::performOpticalFlow()
{
    cv::Mat frame = overheadCam.getFrame();
    cv::Mat drawingImage = frame.clone();

    // upload to gpu + convert to grayscale
    cv::cuda::GpuMat gpuFrame;
    cv::cuda::GpuMat gpuGrayFrame;
    gpuFrame.upload(frame);
    cv::cuda::cvtColor(gpuFrame, gpuGrayFrame, cv::COLOR_BGR2GRAY);


    if (!isInitialized)
    {
        opticalFlow.InitializeMotionDetection(gpuGrayFrame);
        isInitialized = true;
    }
    else
    {
        opticalFlow.PerformMotionDetection(gpuGrayFrame, drawingImage);
        cv::imshow("optical flow", drawingImage);
    }
}

bool areMatsEqual(const cv::Mat &mat1, const cv::Mat &mat2)
{
    if (mat1.size() != mat2.size() || mat1.type() != mat2.type())
    {
        // Mats have different sizes or types
        return false;
    }

    // Compute the absolute difference between the current frame and the previous frame
    cv::Mat diff;
    cv::absdiff(mat1, mat2, diff);

    // Convert the difference to grayscale
    cv::Mat grayDiff;
    cv::cvtColor(diff, grayDiff, cv::COLOR_BGR2GRAY);

    return cv::countNonZero(grayDiff) == 0;
}

void Vision::runPipeline()
{
    // get the frame from the camera
    cv::Mat frame = overheadCam.getFrame().clone();


    // Skip the first frame or if the current frame is the same as the previous frame
    if (previousFrame.empty() || areMatsEqual(frame, previousFrame))
    {
        previousFrame = frame;
        return;
    }

    // find the opponent
    cv::Point2f opponent_new = findOpponent(frame, previousFrame);

    // crop the image around the opponent
    unsigned int cropSize = 50;
    cv::Rect opponentRect = cv::Rect(opponent_new.x - cropSize / 2, opponent_new.y - cropSize / 2, cropSize, cropSize);
    // make sure the crop rectangle doesn't go out of bounds
    opponentRect.x = std::max(0, opponentRect.x);
    opponentRect.y = std::max(0, opponentRect.y);
    opponentRect.width = std::min(opponentRect.width, frame.cols - opponentRect.x);
    opponentRect.height = std::min(opponentRect.height, frame.rows - opponentRect.y);

    cv::Mat croppedDrawingImage = frame(opponentRect).clone();
    std::cout << "about to crop. Crop size: " + std::to_string(cropSize) << std::endl;

    cv::cuda::GpuMat gpuFrame;
    cv::cuda::GpuMat gpuGrayFrame;
    gpuFrame.upload(croppedDrawingImage);
    cv::cuda::cvtColor(gpuFrame, gpuGrayFrame, cv::COLOR_BGR2GRAY);
    std::cout << "cropped" << std::endl;

    // if (!isInitialized)
    // {
    //     std::cout << "about to init" << std::endl;
    //     opponentOpticalFlow.InitializeMotionDetection(gpuGrayFrame);
    //     std::cout << "init done" << std::endl;
    //     isInitialized = true;
    // }
    // else
    // {
    //     std::cout << "about to perform motion detection" << std::endl;
    //     opponentOpticalFlow.PerformMotionDetection(gpuGrayFrame, croppedDrawingImage, true, true, false);
    //     std::cout << "motion detection done" << std::endl;
    //     // cv::imshow("optical flow", croppedDrawingImage);
    // }

    cv::waitKey(1);


    // save the current frame
    previousFrame = frame;
}

cv::Size blurSize = cv::Size(14,14);

cv::Point2f Vision::findOpponent(cv::Mat& frame, cv::Mat& previousFrame)
{
    const float MIN_AREA = 1200;

    cv::Point2f center = cv::Point2f(0,0);

    // Compute the absolute difference between the current frame and the previous frame
    cv::Mat diff;
    cv::absdiff(previousFrame, frame, diff);

    // Convert the difference to grayscale
    cv::Mat grayDiff;
    cv::cvtColor(diff, grayDiff, cv::COLOR_BGR2GRAY);

    // Convert the difference to a binary image with a certain threshold    
    cv::Mat thresholdImg;
    cv::threshold(grayDiff, thresholdImg, 50, 255, cv::THRESH_BINARY);

    // blurr and re-thresh to make it more leanient
    cv::blur(thresholdImg, thresholdImg, blurSize);
    cv::threshold(thresholdImg, thresholdImg, 25, 255, cv::THRESH_BINARY);
    cv::imshow("threshold", thresholdImg);

    // find big blobs in the image using a blob detector

    // iterate through every pixel in the image and find the largest blob
    std::vector<cv::Rect> potentialRobots = {};
    for (int y = 0; y < thresholdImg.rows; y += 10)
    {
        for (int x = 0; x < thresholdImg.cols; x += 10)
        {
            // if this pixel is white, then it is part of a blob
            if (thresholdImg.at<uchar>(y, x) == 255)
            {
                // flood fill the blob
                cv::Rect rect;
                // flood fill but don't change the image
                cv::floodFill(thresholdImg, cv::Point(x, y), cv::Scalar(100), &rect);

                // if the blob is larger than the previous largest blob, then update the largest blob
                if (rect.area() > MIN_AREA)
                {
                    // add the rect to the list of potential robots
                    potentialRobots.push_back(rect);
                }
            }
        }
    }

    // for each robot, find the EXACT center of the robot by counting the white pixels in the blob and averaging them
    std::vector<cv::Point2f> robotCenters;
    std::vector<MotionBlob> motionBlobs;
    for (const cv::Rect &rect : potentialRobots)
    {
        // find the average of all the white pixels in the blob
        int numWhitePixels = 0;
        cv::Point2f averageWhitePixel = cv::Point2f(0,0);
        for (int y = rect.y; y < rect.y + rect.height; y++)
        {
            for (int x = rect.x; x < rect.x + rect.width; x++)
            {
                // if this pixel is white, then add it to the average
                if (thresholdImg.at<uchar>(y,x) > 0)
                {
                    averageWhitePixel += cv::Point2f(x,y);
                    numWhitePixels++;
                }
            }
        }
        // divide by the number of white pixels to get the average
        averageWhitePixel /= numWhitePixels;

        // add the average to the list of robot centers
        motionBlobs.emplace_back(MotionBlob{rect, averageWhitePixel, &frame});
    }

    updateRobotTrackers(motionBlobs, frame);

    // clone the frame so we can draw on it
    cv::Mat frameWithCircles = frame.clone();

    cv::Point2f pos1 = robotTrackers[0].getPosition();
    double ang1 = robotTrackers[0].getAngle();
    cv::Point2f pos2 = robotTrackers[1].getPosition();
    double ang2 = robotTrackers[1].getAngle();

    cv::circle(frameWithCircles, pos1, 30, cv::Scalar(255,0,0), 4);
    cv::circle(frameWithCircles, pos2, 30, cv::Scalar(0,0,255), 4);

    // ang1 = angle_wrap(opponentOpticalFlow.GetRotation());

    // std::cout << "velocity: " << robotTrackers[0].getVelocity() << std::endl;
    // if (cv::norm(robotTrackers[0].getVelocity()) > 200)
    // {
    //     double velocity_angle = atan2(robotTrackers[0].getVelocity().y, robotTrackers[0].getVelocity().x);
    //     if (abs(angle_wrap(velocity_angle - ang1)) > M_PI / 2)
    //     {
    //         velocity_angle += M_PI;
    //     }
    //     velocity_angle = angle_wrap(velocity_angle);

    //     if (abs(angle_wrap(velocity_angle - ang1)) > M_PI / 4)
    //     {
    //         ang1 = velocity_angle;
    //         opponentOpticalFlow.SetRotation(velocity_angle);
    //     }
    // }

    // draw lines starting at the center of each robot showing their angles
    const double RADIUS = 30;
    cv::line(frameWithCircles, pos1, pos1 + cv::Point2f(cos(ang1) * RADIUS, sin(ang1) * RADIUS), cv::Scalar(255,0,0), 4);
    cv::line(frameWithCircles, pos2, pos2 + cv::Point2f(cos(ang2) * RADIUS, sin(ang2) * RADIUS), cv::Scalar(0,0,255), 4);

    // scale up th e
    cv::imshow("keypoints", frameWithCircles);

    return pos1;
}


void Vision::updateRobotTrackers(std::vector<MotionBlob>& centers, cv::Mat& frame)
{
    // minimize cost of updating
    if (centers.empty()) return;

    double COST_THRESHOLD = 0.15; // above this cost means we're not updating
    std::vector<bool> updatedTrackers = {};
    // fill updatedTrackers with false
    for (int i = 0; i < robotTrackers.size(); i++) updatedTrackers.push_back(false);

    if (centers.size() == 1)
    {
        double cost1 = robotTrackers[0].getCostOfUpdating(centers[0]);
        double cost2 = robotTrackers[1].getCostOfUpdating(centers[0]);
        std::cout << "cost1: " << cost1 << std::endl;

        // assume the robot is us if we're moving
        if (cost1 < COST_THRESHOLD)
        {
            robotTrackers[0].update(centers[0], frame);
            updatedTrackers[0] = true;
        }
    }
    else
    {
        double cost00 = robotTrackers[0].getCostOfUpdating(centers[0]);
        double cost01 = robotTrackers[0].getCostOfUpdating(centers[1]);
        double cost10 = robotTrackers[1].getCostOfUpdating(centers[0]);
        double cost11 = robotTrackers[1].getCostOfUpdating(centers[1]);

        if (cost00 + cost11 < cost01 + cost10)
        {
            if (cost00 < COST_THRESHOLD)
            {
                robotTrackers[0].update(centers[0], frame);
                updatedTrackers[0] = true;
            }

            if (cost11 < COST_THRESHOLD)
            {
                robotTrackers[1].update(centers[1], frame);
                updatedTrackers[1] = true;
            }
        }
        else
        {
            if (cost01 < COST_THRESHOLD)
            {
                robotTrackers[0].update(centers[1], frame);
                updatedTrackers[0] = true;
            }

            if (cost10 < COST_THRESHOLD)
            {
                robotTrackers[1].update(centers[0], frame);
                updatedTrackers[1] = true;
            }
        }
    }

    // for each tracker that wasn't updated, mark it as invalid
    for (int i = 0; i < robotTrackers.size(); i++)
    {
        if (!updatedTrackers[i])
        {
            robotTrackers[i].invalidate();
        }
    }
}