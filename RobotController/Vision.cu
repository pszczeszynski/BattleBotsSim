#include "hip/hip_runtime.h"
#include "Vision.h"
#include "MathUtils.h"
#include "RobotStateParser.h"

#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudastereo.hpp>

#include <opencv2/calib3d.hpp>
#include <iostream>
#include <opencv2/features2d.hpp>
#include "Graphics/GameLoop.h"

// GLint doesn't have different sizes on different compilers whereas int does
const GLint POINT_CLOUD_WINDOW_WIDTH = 1920, POINT_CLOUD_WINDOW_HEIGHT = 1080;

#define WIDTH 640
#define HEIGHT 480

#define HEIGHT_DISPARITY WIDTH * 0.6

#define DISPARITY_SCALAR 3500.0

// camera properties obtained from unity
#define FOV_X TO_RAD * 75.18 * 2
#define FOV_Y TO_RAD * 60.0 * 2

const int NUM_DISPARITIES = 256;
const int MIN_DISPARITY = 0;
const int BLOCK_SIZE = 5;
const int MAX_POINT_CLOUD_SIZE = 50000;

Vision::Vision(CameraReceiver &cameraTL, CameraReceiver &cameraTR, CameraReceiver &cameraBL, CameraReceiver &cameraBR,
               CameraReceiver &cameraLL, CameraReceiver &cameraLR, CameraReceiver &cameraRL, CameraReceiver &cameraRR)
    : cameraTL(cameraTL),
      cameraTR(cameraTR),
      cameraBL(cameraBL),
      cameraBR(cameraBR),
      cameraLL(cameraLL),
      cameraLR(cameraLR),
      cameraRL(cameraRL),
      cameraRR(cameraRR)
{
    // Setup StereoSGBM
    const int channels = 1;
    stereoSGMMain = cv::cuda::createStereoSGM(MIN_DISPARITY, NUM_DISPARITIES, 18, 120, 5, 3);
    // Define the stereo matching method and parameters
    stereoSGMMain->setBlockSize(BLOCK_SIZE);
    // stereoSGMMain->setBlockSize(BLOCK_SIZE);
    stereoSGMMain->setP1(2 * channels * 3 * 3);
    stereoSGMMain->setP2(4 * channels * 3 * 3); // increasing makes sortof smoother -> more blobby

    gameLoopThread = new std::thread([this]()
                                     {
        // setup the window
        const Engine::WindowSettings myWindowSettings = {POINT_CLOUD_WINDOW_WIDTH, POINT_CLOUD_WINDOW_HEIGHT};
        Engine::Window myWindow = Engine::Window(myWindowSettings);
        // show the window
        myWindow.Show();

        // this will handle all the logic of our game
        GameLoop gameLoop(WIDTH, HEIGHT, &myWindow);
        pGameLoop = &gameLoop;

        // bind gameLoop's update function to the window
        myWindow.addLoopFunction([&gameLoop]()
                                { gameLoop.update(); });
        myWindow.addInitFunction([&gameLoop]()
                                { gameLoop.init(); });
        // this will start calling the gameLoop update and also polling events
        myWindow.startLoop(); });
}

void extendImageLeftSide(const cv::Mat &src, cv::Mat &dst, int width)
{
    // Create an image with the added black rectangle
    dst = cv::Mat(src.rows, src.cols + width, CV_8UC3, cv::Scalar(0, 0, 0));
    src.copyTo(dst(cv::Rect(width, 0, src.cols, src.rows)));
}

void Vision::computeDisparity(const cv::Mat &left, const cv::Mat &right, cv::Mat &disparity)
{
    cv::Mat leftCrop = left(cv::Rect(cv::Point2f(0, 0), cv::Point2f(WIDTH, HEIGHT_DISPARITY)));
    cv::Mat rightCrop = right(cv::Rect(cv::Point2f(0, 0), cv::Point2f(WIDTH, HEIGHT_DISPARITY)));

    // extend the left side of the images with black. This is because the greater
    // NUM_DISPARITIES, the greater crop on the left side the disparity map will have.
    cv::Mat leftCropExtended;
    extendImageLeftSide(leftCrop, leftCropExtended, NUM_DISPARITIES);
    cv::Mat rightCropExtended;
    extendImageLeftSide(rightCrop, rightCropExtended, NUM_DISPARITIES);

    cv::Mat leftChannels[3], rightChannels[3];
    cv::split(leftCropExtended, leftChannels);
    cv::split(rightCropExtended, rightChannels);

    cv::cuda::GpuMat d_disparityChannels[3];

    // Compute the disparity map for each color channel
    for (int i = 0; i < 3; i++)
    {
        cv::cuda::GpuMat d_leftChannel(leftChannels[i]);
        cv::cuda::GpuMat d_rightChannel(rightChannels[i]);
        stereoSGMMain->compute(d_leftChannel, d_rightChannel, d_disparityChannels[i]);
        d_leftChannel.release();
        d_rightChannel.release();
    }

    // Max the disparity maps for each color channel
    // Find the maximum disparity value for each pixel across all channels
    cv::cuda::GpuMat d_disparity_max(d_disparityChannels[0].size(), d_disparityChannels[0].type(), cv::Scalar(0));
    for (int i = 0; i < 3; i++)
    {
        cv::cuda::max(d_disparity_max, d_disparityChannels[i], d_disparity_max);
        d_disparityChannels[i].release();
    }

    d_disparity_max.download(disparity);
    d_disparity_max.release();

    // Crop the left side of the image
    disparity = disparity(cv::Rect(NUM_DISPARITIES, 0, disparity.cols - NUM_DISPARITIES, disparity.rows));
    // medianBlur(disparity, disparity, 5);

    cv::Mat disparityNormalized;
    // Normalize the disparity map
    cv::normalize(disparity, disparityNormalized, 0, 255, cv::NORM_MINMAX, CV_8U);

    disparityNormalized = 255 - disparityNormalized;
}

cv::Point3f Vision::convert2dPointTo3d(int x, int y, short disparity)
{
    // calc normalized positions
    // 1 means fully to the top of the image or to the right
    // -1 means fully to the bottom of the image or to the left
    float xNormalized = (x - (WIDTH / 2.0)) / (WIDTH / 2.0);
    float yNormalized = -(y - (HEIGHT / 2.0)) / (HEIGHT / 2.0);
    float zPos = DISPARITY_SCALAR / (disparity);

    cv::Point3f point{xNormalized, yNormalized, zPos * 1.3f};

    point.x = xNormalized * sin(FOV_X / 2) * zPos;
    point.y = yNormalized * sin(FOV_Y / 2) * zPos;

    return point;
}

cv::Point3f Vision::rotatePointToRobotSide(cv::Point3f p, Vision::RobotSide robotSide)
{
    switch (robotSide)
    {
    case RobotSide::Front:
        // do nothing
        break;

    case RobotSide::Back:
        p.z *= -1;
        p.x *= -1;
        break;

    case RobotSide::Right:
        std::swap(p.x, p.z);
        p.z *= -1;
        break;

    case RobotSide::Left:
        std::swap(p.x, p.z);
        p.x *= -1;

        break;
    }

    return p;
}

void Vision::compute3dPointCloud(cv::Mat &leftCam, cv::Mat &rightCam,
                                 std::vector<cv::Point3f> &pointCloud, std::vector<cv::Vec3b> &colors, Vision::RobotSide robotSide)
{
    cv::Mat disparity;
    computeDisparity(leftCam, rightCam, disparity);

    for (int y = 0; y < disparity.rows; y += 1)
    {
        for (int x = 1; x < disparity.cols - 1; x += 1)
        {
            short disparityMid = disparity.at<short>(y, x);
            short disparityRight = disparity.at<short>(y, x + 1);
            short disparityLeft = disparity.at<short>(y, x - 1);
            
            short usedDisparity = disparityMid;
            if (disparityRight > usedDisparity) usedDisparity = disparityRight;
            if (disparityLeft > usedDisparity) usedDisparity = disparityLeft;

            // only send pixels with disparities greater than 0
            if (usedDisparity > 0)
            {
                cv::Point3f p = convert2dPointTo3d(x, y, usedDisparity);
                if (p.y > 5 || cv::norm(p) > 16 || p.y < -0.5)
                {
                    continue;
                }
                colors.push_back(leftCam.at<cv::Vec3b>(y, x));
                // rotate the point depending on which side these cameras are on
                p = rotatePointToRobotSide(p, robotSide);
                pointCloud.push_back(p);

                if (pointCloud.size() > MAX_POINT_CLOUD_SIZE)
                {
                    break;
                }
            }
        }
    }
    leftCam.release();
    rightCam.release();
}

struct Cluster
{
    std::vector<cv::Point3f> points;
    cv::Point3f center;

    cv::Point3f boundingBoxMin; // holds the max x y and z in the cluster
    cv::Point3f boundingBoxMax; // holds the min x y and z in the cluster

    Cluster(cv::Point3f initPoint)
    {
        points.push_back(initPoint);
        boundingBoxMax = initPoint;
        boundingBoxMin = initPoint;
        center = initPoint;
    }

    // adds a point to the cluster and updates internal fields
    void AddPoint(cv::Point3f p)
    {
        points.push_back(p);
        center = (center * (float)points.size() + p) / (float)(points.size() + 1);

        // update maxes
        boundingBoxMax.x = max(boundingBoxMax.x, p.x);
        boundingBoxMax.y = max(boundingBoxMax.y, p.y);
        boundingBoxMax.z = max(boundingBoxMax.z, p.z);

        // update mins
        boundingBoxMin.x = min(boundingBoxMin.x, p.x);
        boundingBoxMin.y = min(boundingBoxMin.y, p.y);
        boundingBoxMin.z = min(boundingBoxMin.z, p.z);
    }
};

// Compute the similarity score between a given color and yellow, on a scale from 0 to 1
double getSimilarityScore(cv::Vec3b color, cv::Vec3b targetColor)
{
    double dist = 0.0;
    for (int i = 0; i < 3; i++)
    {
        dist += pow(color[i] - targetColor[i], 2);
    }
    dist = sqrt(dist);
    double maxDist = sqrt(pow(255, 2) * 3); // The maximum distance possible between two colors in RGB space
    return 1.0 - (dist / maxDist);                                        // Return the score, normalized to a range of 0 to 1
}

Vision::OpponentCandidate Vision::findOpponent(std::vector<cv::Point3f> &pointCloud, std::vector<cv::Vec3b> &colors)
{
    const double THRESHOLD_Y = 0; // the minimum y position for a point to be considered part of the robot
    const int MIN_POINTS = 200;      // the minimum number of points for a cluster to be considered the robot
    const double MAX_DISTANCE = 2;   // the maximum distance between points in a cluster
    const int MAX_NUM_CLUSTERS = 120;
    const double MAX_DISTANCE_TO_ROBOT = 100.0; // if farther away from us than this, ignore

    // find clusters of points above the y threshold
    std::vector<Cluster> clusters;

    for (int i = 0; i < pointCloud.size(); i++)
    {
        cv::Point3f p = pointCloud[i];

        const cv::Vec3b YELLOW = cv::Vec3b(0, 101, 188);
        double similarityToYellow = getSimilarityScore(colors[i], YELLOW);
        if (similarityToYellow > 0.75)
        {
            // colors[i] = cv::Vec3b(0, 255, 255);
            continue;
        }
        // skip if below threshold
        if (p.y < THRESHOLD_Y)
        {
            continue;
        }

        bool added = false;
        for (int c = 0; c < clusters.size(); c++)
        {
            if (cv::norm(p - clusters[c].center) <= MAX_DISTANCE)
            {
                // colors[i] = cv::Vec3b((c % 10) / 10.0f * 255.0f, ((c + 3) % 10) / 10.0f * 255.0f, ((c + 6) % 10) / 10.0f * 255.0f);
                clusters[c].AddPoint(p);
                added = true;
                break;
            }
        }

        if (!added)
        {
            // add a new cluster
            clusters.push_back(Cluster{p});

            // break if too many clusters
            if (clusters.size() > MAX_NUM_CLUSTERS)
            {
                break;
            }
        }
    }

    // find the largest cluster that has at least MIN_POINTS points
    OpponentCandidate robotCluster;
    for (int c = 0; c < clusters.size(); c++)
    {
        Cluster &cluster = clusters[c];
        double distanceToRobot = cv::norm(cluster.center);

        // skip if not enough points
        if (cluster.points.size() < MIN_POINTS || distanceToRobot > MAX_DISTANCE_TO_ROBOT)
        {
            continue;
        }

        // compute size of bounding box in each dimension
        cv::Point3f boundSizes = cluster.boundingBoxMax - cluster.boundingBoxMin;
        double maxDimension = boundSizes.x;
        maxDimension = max(maxDimension, boundSizes.y);
        maxDimension = max(maxDimension, boundSizes.z);

        // if this was a cube the size of the maximum dimension, how big would it be?
        double volumeIfWasCube = maxDimension * maxDimension * maxDimension;
        double actualVolume = boundSizes.x * boundSizes.y * boundSizes.z;

        const double IDEAL_VOLUME = 10 * 10 * 10;
        // compute score using size
        double thisScore = 0.25 * (abs(actualVolume - IDEAL_VOLUME) / max(actualVolume, IDEAL_VOLUME)) + 0.5 * (1.0 - cv::norm(cluster.center) / 12.0) + 0.25 * cluster.center.y / 0.5;

        // if beats previous best
        if (thisScore > robotCluster.score)
        {
            // set robot cluster to this
            robotCluster = OpponentCandidate{cluster.center, thisScore};
        }
    }

    return robotCluster;
}

// TODO: don't send opponent position
void Vision::runPipeline(cv::Point3f opponentPositionSim, cv::Point3f motionVector)
{
    // compute individual point clouds
    std::vector<cv::Vec3b> colorsFront;
    std::vector<cv::Point3f> pointCloudFront;
    OpponentCandidate opponentCandidateFront;
    compute3dPointCloud(cameraTL.getFrame(), cameraTR.getFrame(), pointCloudFront, colorsFront, Vision::RobotSide::Front);
    // opponentCandidateFront = findOpponent(pointCloudFront, colorsFront);

    std::vector<cv::Vec3b> colorsBack;
    std::vector<cv::Point3f> pointCloudBack;
    OpponentCandidate opponentCandidateBack;
    compute3dPointCloud(cameraBL.getFrame(), cameraBR.getFrame(), pointCloudBack, colorsBack, Vision::RobotSide::Back);
    // opponentCandidateBack = findOpponent(pointCloudBack, colorsBack);

    std::vector<cv::Vec3b> colorsLeft;
    std::vector<cv::Point3f> pointCloudLeft;
    OpponentCandidate opponentCandidateLeft;
    compute3dPointCloud(cameraLL.getFrame(), cameraLR.getFrame(), pointCloudLeft, colorsLeft, Vision::RobotSide::Left);
    // opponentCandidateLeft = findOpponent(pointCloudLeft, colorsLeft);

    std::vector<cv::Vec3b> colorsRight;
    std::vector<cv::Point3f> pointCloudRight;
    OpponentCandidate opponentCandidateRight;
    compute3dPointCloud(cameraRL.getFrame(), cameraRR.getFrame(), pointCloudRight, colorsRight, Vision::RobotSide::Right);
    // opponentCandidateRight = findOpponent(pointCloudRight, colorsRight);

    // combine point clouds into one
    std::vector<cv::Vec3b> colors;
    std::vector<cv::Point3f> pointCloud;

    pointCloud.insert(pointCloud.end(), pointCloudFront.begin(), pointCloudFront.end());
    colors.insert(colors.end(), colorsFront.begin(), colorsFront.end());

    pointCloud.insert(pointCloud.end(), pointCloudBack.begin(), pointCloudBack.end());
    colors.insert(colors.end(), colorsBack.begin(), colorsBack.end());

    pointCloud.insert(pointCloud.end(), pointCloudLeft.begin(), pointCloudLeft.end());
    colors.insert(colors.end(), colorsLeft.begin(), colorsLeft.end());

    pointCloud.insert(pointCloud.end(), pointCloudRight.begin(), pointCloudRight.end());
    colors.insert(colors.end(), colorsRight.begin(), colorsRight.end());

    // find best opponent position
    OpponentCandidate opponent = {cv::Point3f(0, 0, 0), 0};

    if (opponentCandidateFront.score > opponent.score)
    {
        opponent = opponentCandidateFront;
    }
    if (opponentCandidateBack.score > opponent.score)
    {
        opponent = opponentCandidateBack;
    }
    if (opponentCandidateLeft.score > opponent.score)
    {
        opponent = opponentCandidateLeft;
    }
    if (opponentCandidateRight.score > opponent.score)
    {
        opponent = opponentCandidateRight;
    }

    // if gameloop created, submit vertices + opponent position
    if (pGameLoop)
    {
        pGameLoop->SetPointCloudVerts(pointCloud, colors);
        pGameLoop->SetOpponentPosition({opponentPositionSim.x, opponentPositionSim.y, opponentPositionSim.z});//{opponent.pos.x, 0, opponent.pos.z}
        std::vector<GameLoop::Line> lines = {};
        lines.push_back(GameLoop::Line{cv::Point3f(0, 0, 0),
                                       opponentPositionSim, //{opponent.pos.x, 0, opponent.pos.z}
                                       cv::Vec3b(0, 255, 255)});

        lines.push_back(GameLoop::Line{cv::Point3f(0, 0, 0),
                                       motionVector,
                                       cv::Vec3b(255, 0, 0)});
        pGameLoop->SetPathPlanningLines(lines);
    }
}
